#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

namespace {

template <typename scalar_t>
__global__ void sub_norm_cuda_forward_kernel(
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x1,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x2,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> y) {

  const int n2 = x2.size(2);
  const int n1 = x1.size(2);
  const int c = x1.size(1);

  // // cannot parallize across channels, because it will case modifying the the location by multiple threads at the same time
  // const int inc12 = blockIdx.x * blockDim.x + threadIdx.x;
  
  // const int ic = inc12 / (n1*n2);
  // const int in12 = inc12 % (n1*n2);
  // const int in1 = in12 / n2;
  // const int in2 = in12 % n2;

  // if (inc12 < n1 * n2 * c ){
  //   y[blockIdx.y][in1][in2] += (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]) * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]) ;
  // }

  const int in12 = blockIdx.x * blockDim.x + threadIdx.x;
  
  const int in1 = in12 / n2;
  const int in2 = in12 % n2;
  

  if (in12 < n1 * n2 ){
    for (int ic = 0; ic < c; ic++){
      y[blockIdx.y][in1][in2] += (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]) * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]) ;
    }
  }

}

// template <typename scalar_t>
// __global__ void sub_norm_cuda_backward_kernel(
//   torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx1,
//   torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx2,
//   const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
//   const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x1,
//   const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x2) {

//   const int n2 = dx2.size(2);
//   const int n1 = dx1.size(2);
//   const int c = dx1.size(1);

//   const int inc12 = blockIdx.x * blockDim.x + threadIdx.x;
  
//   const int ic = inc12 / (n1*n2);
//   const int in12 = inc12 % (n1*n2);
//   const int in1 = in12 / n2;
//   const int in2 = in12 % n2;
  
//   if (inc12 < n1 * n2 * c ){
//     dx1[blockIdx.y][ic][in1] += dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
//     dx2[blockIdx.y][ic][in2] -= dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
//   }

// }
// } // namespace

template <typename scalar_t>
__global__ void sub_norm_cuda_backward_kernel_dx(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx1,
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx2,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x1,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x2) {

  const int n2 = x2.size(2);
  const int n1 = x1.size(2);
  const int c = x1.size(1);

  const int inc12 = blockIdx.x * blockDim.x + threadIdx.x;
  const int in12 = inc12 / c;
  const int ic = inc12 % c;

  if (in12 < n1){
    const int in1 = in12;
    if (in1 < n1 ){
      for (int in2 = 0; in2 < n2; in2++){
        dx1[blockIdx.y][ic][in1] += dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
      }
    }
  }
  else{
    const int in2 = in12 - n1;
    if (in2 < n2){
      for (int in1 = 0; in1 < n1; in1++){
        dx2[blockIdx.y][ic][in2] -= dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
      }
    }
  }
}

template <typename scalar_t>
__global__ void sub_norm_cuda_backward_kernel_dx1(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx1,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x1,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x2) {

  const int n2 = x2.size(2);
  const int n1 = x1.size(2);
  const int c = x1.size(1);

  const int inc1 = blockIdx.x * blockDim.x + threadIdx.x;
  
  const int ic = inc1 / n1;
  const int in1 = inc1 % n1;
  
  if (inc1 < n1 * c ){
    for (int in2 = 0; in2 < n2; in2++){
      dx1[blockIdx.y][ic][in1] += dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
    }
  }
}

template <typename scalar_t>
__global__ void sub_norm_cuda_backward_kernel_dx2(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx2,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x1,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> x2) {

  const int n2 = x2.size(2);
  const int n1 = x1.size(2);
  const int c = x1.size(1);

  const int inc2 = blockIdx.x * blockDim.x + threadIdx.x;
  
  const int ic = inc2 / n2;
  const int in2 = inc2 % n2;
  
  if (inc2 < n2 * c ){
    for (int in1 = 0; in1 < n1; in1++){
      dx2[blockIdx.y][ic][in2] -= dy[blockIdx.y][in1][in2] * 2 * (x1[blockIdx.y][ic][in1] - x2[blockIdx.y][ic][in2]);
    }
  }
}
} // namespace

torch::Tensor sub_norm_cuda_forward(
    torch::Tensor x1,
    torch::Tensor x2) {

  const auto batch_size = x1.size(0);
  const auto channel_size = x1.size(1);
  const auto n1 = x1.size(2);
  const auto n2 = x2.size(2);

  auto options = torch::TensorOptions().dtype(x1.dtype()).layout(torch::kStrided).device(x1.device()).requires_grad(true);
  auto y = torch::zeros({batch_size, n1, n2}, options);

  // printf("x1 device: %d \n", x1.device().type()); 
  // printf("x1 index: %d \n", x1.device().index()); 

  const int threads = 1024;
  // cannot parallize across channels, because it will case modifying the the location by multiple threads at the same time
  // const dim3 blocks((n1 * n2 * channel_size + threads - 1) / threads, batch_size);
  const dim3 blocks((n1 * n2  + threads - 1) / threads, batch_size);
  // const dim3 blocks(1, 1);

  int device_id = x1.device().index();
  hipSetDevice(device_id);

  // AT_DISPATCH_FLOATING_TYPES
 AT_DISPATCH_ALL_TYPES_AND_HALF(x1.type(), "sub_norm_forward_cuda", ([&] {
    sub_norm_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
      x1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      x2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      y.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));
  hipDeviceSynchronize();  
  return y;
}

std::vector<torch::Tensor> sub_norm_cuda_backward(
    torch::Tensor dy, 
    torch::Tensor x1,
    torch::Tensor x2) {
  const auto batch_size = dy.size(0);
  const auto channel_size = x1.size(1);
  const auto n1 = dy.size(1);
  const auto n2 = dy.size(2);

  auto dx1 = torch::zeros({batch_size, channel_size, n1}, x1.device());
  auto dx2 = torch::zeros({batch_size, channel_size, n2}, x1.device());

  const int threads = 1024;
  // The backward function can parallize channels because it does not cause same-time modification of the same location
  // But n1 and n2 will conflict with each other when parallelized together 
  // because dx1 (and dx2) may be modified at the same location by multiple thread with the same in1 or in2 index
  // const dim3 blocks((n1 * n2 * channel_size + threads - 1) / threads, batch_size);

  // AT_DISPATCH_FLOATING_TYPES(dy.type(), "sub_norm_forward_cuda", ([&] {
  //   sub_norm_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
  //     dx1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     dx2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
  //     x1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     x2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  // }));

  int device_id = x1.device().index();
  hipSetDevice(device_id);

  const dim3 blocks_dx12(( (n1+n2) * channel_size + threads - 1) / threads, batch_size);
  AT_DISPATCH_ALL_TYPES_AND_HALF(dy.type(), "sub_norm_backward_cuda_dx", ([&] {
    sub_norm_cuda_backward_kernel_dx<scalar_t><<<blocks_dx12, threads>>>(
      dx1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      dx2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
      x1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      x2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));
  hipDeviceSynchronize();  


  // const dim3 blocks_dx1((n1 * channel_size + threads - 1) / threads, batch_size);
  // // AT_DISPATCH_FLOATING_TYPES
  // AT_DISPATCH_ALL_TYPES_AND_HALF(dy.type(), "sub_norm_backward_cuda_dx1", ([&] {
  //   sub_norm_cuda_backward_kernel_dx1<scalar_t><<<blocks_dx1, threads>>>(
  //     dx1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
  //     x1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     x2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  // }));
  // hipDeviceSynchronize();  

  // const dim3 blocks_dx2((n2 * channel_size + threads - 1) / threads, batch_size);

  // // AT_DISPATCH_FLOATING_TYPES
  // AT_DISPATCH_ALL_TYPES_AND_HALF(dy.type(), "sub_norm_backward_cuda_dx2", ([&] {
  //   sub_norm_cuda_backward_kernel_dx2<scalar_t><<<blocks_dx2, threads>>>(
  //     dx2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
  //     x1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
  //     x2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  // }));
  // hipDeviceSynchronize();  

  return {dx1, dx2};
}
