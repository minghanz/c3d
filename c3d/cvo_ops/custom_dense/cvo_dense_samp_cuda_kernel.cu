#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <math.h>

namespace {

template <typename scalar_t>
__global__ void cvo_dense_samp_cuda_forward_kernel(
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts,
    const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts_info,
    const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> grid_source,
    const torch::PackedTensorAccessor<bool,4,torch::RestrictPtrTraits,size_t> grid_valid,
    const int neighbor_range, 
    const float ell,
    const bool ignore_ib, 
    const bool sqr,
    const float ell_basedist, 
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> y) {

  const auto N = pts.size(2);
  const auto C = pts_info.size(1);
  const auto B = grid_source.size(0);
  const auto H = grid_source.size(2);
  const auto W = grid_source.size(3);
  const int NN_sqrt = 2 * neighbor_range + 1;

  //dim3 block[N, NN, 1]
  const auto in = blockIdx.x * blockDim.x + threadIdx.x;  
  const int innh = blockIdx.y / NN_sqrt - neighbor_range;
  const int innw = blockIdx.y % NN_sqrt - neighbor_range;

  if (in < N ){
    const int u = pts[0][0][in];
    const int v = pts[0][1][in];
    int ib;
    if (ignore_ib){
      ib = 0;
    }
    else{
      ib = pts[0][2][in];
    }
    if (u+innw >= 0 && u+innw < W && v+innh >= 0 && v+innh < H){
      if (grid_valid[ib][0][v+innh][u+innw] > 0){

        float ell_apply;
        if (ell_basedist!= 0){
          float flat_z = pts_info[0][2][in];
          ell_apply = max(flat_z, ell_basedist) / ell_basedist * ell;
        }
        else{
          ell_apply = ell;
        }

        float d_cur = 0;
        for (int ic = 0; ic < C; ic++){
          d_cur += (pts_info[0][ic][in] - grid_source[ib][ic][v+innh][u+innw]) * (pts_info[0][ic][in] - grid_source[ib][ic][v+innh][u+innw]);
        }
        if (sqr){
          y[0][blockIdx.y][in] = exp( - d_cur / (2*ell_apply*ell_apply) ) ;
        }
        else{
          y[0][blockIdx.y][in] = exp( - sqrt(d_cur+1e-8) / ell_apply ) ;
        }
        
      }
    }
  }

}


template <typename scalar_t>
__global__ void cvo_dense_samp_cuda_backward_kernel_sqr_only(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx1,
  torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> dx2,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> y, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts_info,
  const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> grid_source,
  const torch::PackedTensorAccessor<bool,4,torch::RestrictPtrTraits,size_t> grid_valid,
  const int neighbor_range, 
  const float ell, 
  const bool ignore_ib, 
  const bool sqr,
  const float ell_basedist, 
  const int inn) {
  // dx1: 1*C*N
  // dx2: B*C*H*W
  // dy: 1*NN*N

  const auto N = pts.size(2);
  const auto C = pts_info.size(1);
  const auto B = grid_source.size(0);
  const auto H = grid_source.size(2);
  const auto W = grid_source.size(3);
  const auto NN = (2*neighbor_range+1)*(2*neighbor_range+1);
  const auto NN_sqrt = 2 * neighbor_range + 1;

  //dim3 block[N, C, 1] 
  if (inn < NN){
    const int in = blockIdx.x * blockDim.x + threadIdx.x;
    const int innh = inn / NN_sqrt - neighbor_range;
    const int innw = inn % NN_sqrt - neighbor_range;

    if (in < N ){
      const int u = pts[0][0][in];
      const int v = pts[0][1][in];
      int ib;
      if (ignore_ib){
        ib = 0;
      }
      else{
        ib = pts[0][2][in];
      }
      if (u+innw >= 0 && u+innw < W && v+innh >= 0 && v+innh < H){
        if (grid_valid[ib][0][v+innh][u+innw] > 0){

          float ell_apply;
          if (ell_basedist!= 0){
            float flat_z = pts_info[0][2][in];
            ell_apply = max(flat_z, ell_basedist) / ell_basedist * ell;
          }
          else{
            ell_apply = ell;
          }

          dx1[0][blockIdx.y][in] += dy[0][inn][in] * y[0][inn][in] * (grid_source[ib][blockIdx.y][v+innh][u+innw] - pts_info[0][blockIdx.y][in]) / (ell_apply*ell_apply);
          dx2[ib][blockIdx.y][v+innh][u+innw] -= dy[0][inn][in] * y[0][inn][in] * (grid_source[ib][blockIdx.y][v+innh][u+innw] - pts_info[0][blockIdx.y][in]) / (ell_apply*ell_apply);
        }
      }
    }
  }
  
}

template <typename scalar_t>
__global__ void cvo_dense_samp_cuda_backward_kernel(
  torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dx1,
  torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> dx2,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> dy, 
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts,
  const torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> pts_info,
  const torch::PackedTensorAccessor<scalar_t,4,torch::RestrictPtrTraits,size_t> grid_source,
  const torch::PackedTensorAccessor<bool,4,torch::RestrictPtrTraits,size_t> grid_valid,
  const int neighbor_range, 
  const float ell, 
  const bool ignore_ib, 
  const bool sqr,
  const float ell_basedist, 
  const int inn) {
  // dx1: 1*C*N
  // dx2: B*C*H*W
  // dy: 1*NN*N

  const auto N = pts.size(2);
  const auto C = pts_info.size(1);
  const auto B = grid_source.size(0);
  const auto H = grid_source.size(2);
  const auto W = grid_source.size(3);
  const auto NN = (2*neighbor_range+1)*(2*neighbor_range+1);
  const auto NN_sqrt = 2 * neighbor_range + 1;

  //dim3 block[N, C, 1] 
  if (inn < NN){
    const int in = blockIdx.x * blockDim.x + threadIdx.x;
    const int innh = inn / NN_sqrt - neighbor_range;
    const int innw = inn % NN_sqrt - neighbor_range;

    if (in < N ){
      const int u = pts[0][0][in];
      const int v = pts[0][1][in];
      int ib;
      if (ignore_ib){
        ib = 0;
      }
      else{
        ib = pts[0][2][in];
      }
      if (u+innw >= 0 && u+innw < W && v+innh >= 0 && v+innh < H){
        if (grid_valid[ib][0][v+innh][u+innw] > 0){

          float ell_apply;
          if (ell_basedist!= 0){
            float flat_z = pts_info[0][2][in];
            ell_apply = max(flat_z, ell_basedist) / ell_basedist * ell;
          }
          else{
            ell_apply = ell;
          }

          float d_cur = 0;
          for (int ic = 0; ic < C; ic++){
            d_cur += (pts_info[0][ic][in] - grid_source[ib][ic][v+innh][u+innw]) * (pts_info[0][ic][in] - grid_source[ib][ic][v+innh][u+innw]);
          }

          float y_cur;
          if (sqr){
            y_cur = exp( - d_cur / (2*ell_apply*ell_apply) );
            for (int ic = 0; ic < C; ic++){
              dx1[0][ic][in] += dy[0][inn][in] * y_cur * (grid_source[ib][ic][v+innh][u+innw] - pts_info[0][ic][in]) / (ell_apply*ell_apply);
              dx2[ib][ic][v+innh][u+innw] -= dy[0][inn][in] * y_cur * (grid_source[ib][ic][v+innh][u+innw] - pts_info[0][ic][in]) / (ell_apply*ell_apply);
            }
          }
          else{
            d_cur = sqrt(d_cur + 1e-8);
            y_cur = exp( - d_cur / ell_apply );
            for (int ic = 0; ic < C; ic++){
              dx1[0][ic][in] += dy[0][inn][in] * y_cur * (grid_source[ib][ic][v+innh][u+innw] - pts_info[0][ic][in]) / ell_apply / d_cur;
              dx2[ib][ic][v+innh][u+innw] -= dy[0][inn][in] * y_cur * (grid_source[ib][ic][v+innh][u+innw] - pts_info[0][ic][in]) / ell_apply / d_cur;
            }

          }
        }
      }
    }
  }
  
}

} // namespace

torch::Tensor cvo_dense_samp_cuda_forward(
    torch::Tensor pts,
    torch::Tensor pts_info, 
    torch::Tensor grid_source, 
    torch::Tensor grid_valid, 
    int neighbor_range,
    float ell, 
    bool ignore_ib, 
    bool sqr,
    float ell_basedist
    ) {
    // pts: 1*2*N, pts_info: 1*C*N, grid_source: B*C*H*W (C could be xyz, rgb, ...), 
    // grid_valid: B*1*H*W, neighbor_range: int

  const auto N = pts.size(2);
  const auto C = pts_info.size(1);
  const auto B = grid_source.size(0);
  const auto H = grid_source.size(2);
  const auto W = grid_source.size(3);
  const auto NN = (2*neighbor_range+1)*(2*neighbor_range+1);

  auto options = torch::TensorOptions().dtype(pts_info.dtype()).layout(torch::kStrided).device(pts_info.device()).requires_grad(true);
  auto y = torch::zeros({1, NN, N}, options);

  // printf("x1 device: %d \n", x1.device().type()); 
  // printf("x1 index: %d \n", x1.device().index()); 

  const int threads = 1024;
  // cannot parallize across channels, because it will case modifying the the location by multiple threads at the same time
  // const dim3 blocks((n1 * n2 * channel_size + threads - 1) / threads, batch_size);
  const dim3 blocks((N  + threads - 1) / threads, NN);
  // const dim3 blocks(1, 1);

  int device_id = pts_info.device().index();
  hipSetDevice(device_id);

  // AT_DISPATCH_FLOATING_TYPES // AT_DISPATCH_ALL_TYPES_AND_HALF
  AT_DISPATCH_FLOATING_TYPES(pts_info.type(), "cvo_dense_samp_forward_cuda", ([&] {
    cvo_dense_samp_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
      pts.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      pts_info.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
      grid_source.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
      grid_valid.packed_accessor<bool,4,torch::RestrictPtrTraits,size_t>(),
      neighbor_range, 
      ell,
      ignore_ib,
      sqr,
      ell_basedist, 
      y.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));
  hipDeviceSynchronize();


  return y;
}

std::vector<torch::Tensor> cvo_dense_samp_cuda_backward(
    torch::Tensor dy, 
    torch::Tensor pts,
    torch::Tensor pts_info, 
    torch::Tensor grid_source, 
    torch::Tensor grid_valid, 
    int neighbor_range,
    float ell, 
    bool ignore_ib, 
    bool sqr,
    float ell_basedist
    ) {

  // dy: 1*NN*N

  const auto N = pts.size(2);
  const auto C = pts_info.size(1);
  const auto B = grid_source.size(0);
  const auto H = grid_source.size(2);
  const auto W = grid_source.size(3);
  const auto NN = (2*neighbor_range+1)*(2*neighbor_range+1);

  auto dx1 = torch::zeros({1, C, N}, pts_info.device());
  auto dx2 = torch::zeros({B, C, H, W}, pts_info.device());

  const int threads = 1024;

  int device_id = pts_info.device().index();
  hipSetDevice(device_id);

  // const dim3 blocks_dx12(( N + threads - 1) / threads, C); // for cvo_dense_samp_cuda_backward_kernel_sqr_only, need y
  const dim3 blocks_dx12(( N + threads - 1) / threads);

  for (int inn = 0; inn < NN; inn++){
    // AT_DISPATCH_FLOATING_TYPES(dy.type(), "cvo_dense_samp_backward_cuda_dx", ([&] {
    //   cvo_dense_samp_cuda_backward_kernel_sqr_only<scalar_t><<<blocks_dx12, threads>>>(
    //     dx1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
    //     dx2.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
    //     dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
    //     y.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
    //     pts.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
    //     pts_info.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
    //     grid_source.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
    //     grid_valid.packed_accessor<bool,4,torch::RestrictPtrTraits,size_t>(),
    //     neighbor_range, 
    //     ell, 
    //     ignore_ib, 
    //     sqr,
    //     ell_basedist, 
    //     inn);
    // }));
    AT_DISPATCH_FLOATING_TYPES(dy.type(), "cvo_dense_samp_backward_cuda", ([&] {
      cvo_dense_samp_cuda_backward_kernel<scalar_t><<<blocks_dx12, threads>>>(
        dx1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        dx2.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        dy.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(), 
        pts.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        pts_info.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        grid_source.packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        grid_valid.packed_accessor<bool,4,torch::RestrictPtrTraits,size_t>(),
        neighbor_range, 
        ell, 
        ignore_ib, 
        sqr,
        ell_basedist, 
        inn);
    }));
    hipDeviceSynchronize();  
  }

  return {dx1, dx2};
}
